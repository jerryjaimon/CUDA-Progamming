#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "Inversion_CUDA.h"
#include "stdio.h";
__global__ void Inversion_CUDA(unsigned char* Image, int Channels);


void Image_Inversion_CUDA(unsigned char* Input_Image, int Height, int Width, int Channels) {
	unsigned char* Dev_Input_Image = NULL;
	hipMalloc((void**)&Dev_Input_Image, Height * Width * Channels);
	hipMemcpy(Dev_Input_Image, Input_Image, Height * Width * Channels, hipMemcpyHostToDevice);
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	dim3 Grid_Image(Width, Height);

	hipEventRecord(start); {
	Inversion_CUDA << <Grid_Image, 1 >> > (Dev_Input_Image, Channels);
	}
	hipEventRecord(stop);
	float ms = 0.0f;
	hipEventSynchronize(stop);
	hipEventElapsedTime(&ms, start, stop);
	printf("\n\tElapsed GPU time : %fms", ms);

	hipMemcpy(Input_Image, Dev_Input_Image, Height * Width * Channels, hipMemcpyDeviceToHost);
	hipFree(Dev_Input_Image);

	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
	}
}

__global__ void Inversion_CUDA(unsigned char* Image, int Channels) {
	int x = blockIdx.x;
	int y = blockIdx.y;
	int idx = (x + y * gridDim.x) * Channels;

	for (int i = 0; i < Channels; i++) {
		Image[idx + i] = 255 - Image[idx + i];
	}
}
